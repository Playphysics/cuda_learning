#include <hip/hip_runtime.h>
#include <stdio.h>

static void print8Data(const float *ptr) {
    for (int i = 0; i < 8; ++i) {
        printf("%10.5f", ptr[i]);
    }
    printf("\n");
}

#define CHECK(funcExpr)                                            \
    do {                                                           \
        const hipError_t errCode = funcExpr;                      \
        if (errCode != hipSuccess) {                              \
            printf("\n=== ERROR ====== ERROR ====== ERROR ===\n"); \
            printf("line: %d %s\n", __LINE__, #funcExpr);          \
            printf("%s\n\n", hipGetErrorString(errCode));         \
        }                                                          \
    } while (0)

#define MAX_NUM 32U

__global__ void ArrayAdd(float *dest, const float *arrA, const float *arrB) {
    const size_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < MAX_NUM) {
        dest[idx] = arrA[idx] + arrB[idx];
    }
}

int main(int argc, char **argv) {
    float *arrA, *arrB, *arrC;
    const size_t nBytes = sizeof(float) * MAX_NUM;

    CHECK(hipSetDevice(0));

    CHECK(hipMallocManaged(&arrA, nBytes));
    CHECK(hipMallocManaged(&arrB, nBytes));
    CHECK(hipMallocManaged(&arrC, nBytes));

    for (int i = 0; i < MAX_NUM; ++i) {
        arrA[i] = i * 1.2f;
        arrB[i] = i / 2.0f;
    }

    dim3 block(MAX_NUM, 1), grid(1, 1);

    ArrayAdd<<<grid, block>>>(arrC, arrA, arrB);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    print8Data(&arrA[0]), print8Data(&arrA[8]), putchar('\n');
    print8Data(&arrB[0]), print8Data(&arrB[8]), putchar('\n');
    print8Data(&arrC[0]), print8Data(&arrC[8]), putchar('\n');

    CHECK(hipFree(arrA));
    CHECK(hipFree(arrB));
    CHECK(hipFree(arrC));

    CHECK(hipDeviceReset());
    return 0;
}
